#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------

// Torch port:
// IMAGINE, Sergey Zagoruyko, Francisco Massa, 2015

#include "THC.h"
#include <algorithm>
#include <cfloat>
#include "assert.h"

#include "common.h"


using std::max;
using std::min;


template <typename Dtype>
__global__ void ROIWarpForward(const int nthreads, const Dtype* bottom_data,
    const Dtype spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois, const Dtype* bottom_delta_rois, Dtype* top_data/*, int* argmax_data*/) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = (bottom_rois[0] - 1);
    //int roi_start_w = round((bottom_rois[1] - 1) * spatial_scale);
    //int roi_start_h = round((bottom_rois[2] - 1)* spatial_scale);
    //int roi_end_w = round((bottom_rois[3] - 1) * spatial_scale);
    //int roi_end_h = round((bottom_rois[4] - 1) * spatial_scale);

    Dtype src_w = bottom_rois[3] - bottom_rois[1] + 1; 
    Dtype src_h = bottom_rois[4] - bottom_rois[2] + 1;
    Dtype src_ctr_x = bottom_rois[1] + 0.5*(src_w-1.0); 
    Dtype src_ctr_y = bottom_rois[2] + 0.5*(src_h-1.0); 

    Dtype dst_ctr_x = bottom_delta_rois[1]; // dx (in fast-rcnn notation) = cx (in here)
    Dtype dst_ctr_y = bottom_delta_rois[2]; // dy (in fast-rcnn notation) = cy (in here) 
    Dtype dst_scl_x = bottom_delta_rois[3]; // dw (in fast-rcnn notation) = sx (in here)
    Dtype dst_scl_y = bottom_delta_rois[4]; // dh (in fast-rcnn notation) = sy (in here) 

    Dtype pred_ctr_x = dst_ctr_x * src_w + src_ctr_x; // dpcx / dcx = src_w
    Dtype pred_ctr_y = dst_ctr_y * src_h + src_ctr_y; // dpcy / dcy = src_h
    Dtype pred_w = exp(dst_scl_x) * src_w;            // dpw  / dsx = src_w * exp(dsx)  
    Dtype pred_h = exp(dst_scl_y) * src_h;            // dph  / dsy = src_h * exp(dsy)  
    
    int roi_start_w = ( round(pred_ctr_x - 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drsw / dpcx = spatial_scale; drsw / dpw = -0.5 * spatial_scale
    int roi_start_h = ( round(pred_ctr_y - 0.5*(pred_h-1)) - 1 ) * spatial_scale; // drsh / dpcy = spatial_scale; drsh / dph = -0.5 * spatial_scale
    int roi_end_w =   ( round(pred_ctr_x + 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drew / dpcx = spatial_scale; drew / dpw =  0.5 * spatial_scale
    int roi_end_h =   ( round(pred_ctr_y + 0.5*(pred_h-1)) - 1 ) * spatial_scale; // dreh / dpcy = spatial_scale; dreh / dph =  0.5 * spatial_scale
    assert(roi_end_w - roi_start_w >= 0);
    assert(roi_end_h - roi_start_h >= 0);   
    
    // drsw / dcx = drsw / dpcx * dpcx / dcx = spatial_scale * src_w
    // drew / dcx = drew / dpcx * dpcx / dcx = spatial_scale * src_w

    // drsh / dcy = drsh / dpcy * dpcy / dcy = spatial_scale * src_h
    // dreh / dcy = dreh / dpcy * dpcy / dcy = spatial_scale * src_h

    // drsw / dsx = drsw / dpw * dpw / dsx = -0.5 * spatial_scale * src_w * exp(dsx) 
    // drew / dsx = drew / dpw * dpw / dsx =  0.5 * spatial_scale * src_w * exp(dsx)
 
    // drsh / dsy = drsh / dph * dph / dsy = -0.5 * spatial_scale * src_h * exp(dsy)
    // dreh / dsy = dreh / dph * dph / dsy =  0.5 * spatial_scale * src_h * exp(dsy) 
 
    // Force malformed ROIs to be 1x1
    int roi_width  = roi_end_w - roi_start_w + 1;    // drw / drew = (rew - rsw) > 0 ?  1 : 0 
                                                    // drw / drsw = (rew - rsw) > 0 ? -1 : 0
    int roi_height = roi_end_h - roi_start_h + 1;   // drh / dreh = (reh - rsh) > 0 ?  1 : 0
                                                    // drh / drsh = (reh - rsh) > 0 ? -1 : 0
    // drw / dcx = drw / drew * drew / dcx + drw / drsw * drsw / dcx = drew / dcx - drsw / dcx = spatial_scale * src_w - spatial_scale * src_w = 0 
    // drh / dcy = drh / dreh * dreh / dcy + drh / drsh * drsh / dcy = dreh / dcy - drsh / dcy = spatial_scale * src_h - spatial_scale * src_h = 0 
    // drw / dsx = drw / drew * drew / dsx + drw / drsw * drsw / dsx = drew / dsx - drsw / dsx = 0.5 * spatial_scale * src_w * exp(dsx) - (-0.5 * spatial_scale * src_w * exp(dsx)) = spatial_scale * src_w * exp(dsx) 
    // drh / dsy = drh / dreh * dreh / dsy + drh / drsh * drsh / dsy = dreh / dsy - drsh / dsy = 0.5 * spatial_scale * src_h * exp(dsy) - (-0.5 * spatial_scale * src_h * exp(dsy)) = spatial_scale * src_h * exp(dsy) 
    
    Dtype bin_size_w = static_cast<Dtype>(roi_width)
                       / static_cast<Dtype>(pooled_width);  // dbw / drw  =  1 / pooled_width
    Dtype bin_size_h = static_cast<Dtype>(roi_height)         
                       / static_cast<Dtype>(pooled_height); // dbh / drh  =  1 / pooled_height
    // dbw / dcx = dbw / drw * drw / dcx = 0 
    // dbh / dcy = dbh / drh * drh / dcy = 0
    // dbw / dsx = dbw / drw * drw / dsx = 1 / pooled_width * spatial_scale * src_w * exp(dsx) 
    // dbh / dsy = dbh / drh * drh / dsy = 1 / pooled_height * spatial_scale * src_h * exp(dsy) 

    int wstart = static_cast<int>(floor(static_cast<Dtype>(pw)        // dws / dbw = pw 
                                        * bin_size_w)) + roi_start_w; // dws / drsw = 1
    int hstart = static_cast<int>(floor(static_cast<Dtype>(ph)        // dhs / dbh = ph 
                                        * bin_size_h)) + roi_start_h; // dhs / drsh = 1 
    int wend = static_cast<int>(ceil(static_cast<Dtype>(pw + 1)       // dwe / dbw = (pw+1)
                                     * bin_size_w)) + roi_start_w;    // dwe / drsw = 1 
    int hend = static_cast<int>(ceil(static_cast<Dtype>(ph + 1)       // dhe / dbh = (ph+1)
                                     * bin_size_h)) + roi_start_h;    // dhe / drsh = 1 
    // dws / dcx = dws / dbw * dbw / dcx + dws / drsw * drsw / dcx = pw * 0 + 1 * spatial_scale * src_w     = spatial_scale * src_w
    // dwe / dcx = dwe / dbw * dbw / dcx + dwe / drsw * drsw / dcx = (pw+1) * 0 + 1 * spatial_scale * src_w = spatial_scale * src_w

    // dws / dsx = dws / dbw * dbw / dsx + dws / drsw * drsw / dsx = pw * 1 / pooled_width * spatial_scale * src_w * exp(dsx) + 1 * (-0.5) * spatial_scale * src_w * exp(dsx) = ( pw / pooled_width - 0.5 ) * spatial_scale * src_w * exp(dsx) 
    // dwe / dsx = dwe / dbw * dbw / dsx + dwe / drsw * drsw / dsx = (pw+1) * 1 / pooled_width * spatial_scale * src_w * exp(dsx) + 1 * 0.5 * spatial_scale * src_w * exp(dsx) = ( (pw+1)/pooled_width + 0.5 ) * spatial_scale * src_w * exp(dsx)

    // dhs / dcy = dhs / dbh * dbh / dcy + dhs / drsh * drsh / dcy = ph * 0 + 1 * spatial_scale * src_h     = spatial_scale * src_w
    // dhe / dcy = dhe / dbh * dbh / dcy + dhe / drsh * drsh / dcy = (ph+1) * 0 + 1 * spatial_scale * src_h = spatial_scale * src_h

    // dhs / dsy = dhs / dbh * dbh / dsy + dhs / drsh * drsh / dsy = ph * 1 / pooled_height * spatial_scale * src_h * exp(dsy) + 1 * (-0.5) * spatial_scale * src_h * exp(dsy) = (ph / pooled_height - 0.5) * spatial_scale * src_h * exp(dsy) 
    // dhe / dsy = dhe / dbh * dbh / dsy + dhe / drsh * drsh / dsy = (ph+1) * 1 / pooled_height * spatial_scale * src_h * exp(dsy) + 1 * 0.5 * spatial_scale * src_h * exp(dsy) = ((ph+1)/pooled_height + 0.5) * spatial_scale * src_h * exp(dsy)  

    Dtype wctr = static_cast<Dtype>(wend-1+wstart) * 0.5;    // dwctr / dwe = 0.5; dwctr / dws = 0.5 
    Dtype hctr = static_cast<Dtype>(hend-1+hstart) * 0.5;    // dhctr / dhe = 0.5; dhctr / dhs = 0.5 
    Dtype wdiff = max(static_cast<Dtype>(wend-1-wstart), 1.);         // dwdiff / dwe = 1; dwdiff / dws = -1
    Dtype hdiff = max(static_cast<Dtype>(hend-1-hstart), 1.);         // dhdiff / dhe = 1; dhdiff / dhs = -1
    // dwctr / dcx = dwctr / dwe * dwe / dcx + dwctr / dws * dws / dcx = 0.5 * spatial_scale * src_w + 0.5 * spatial_scale * src_w = spatial_scale * src_w 
    // dwdiff / dcx = dwdiff / dwe * dwe / dcx + dwdiff / dws * dws / dcx = 1 * spatial_scale * src_w -  1  * spatial_scale * src_w = 0 

    // dhctr / dcy = spatial_scale * src_h
    // dhdiff / dcy = 0
  
    // dwctr / dsx = dwctr / dwe * dwe / dsx + dwctr / dws * dws / dsx = 0.5 * ((pw+1)/pooled_width + 0.5) * spatial_scale * src_w * exp(dsx) + 0.5 * (pw/pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)
    //                                                                 = 0.5 * (2*pw+1)/pooled_width * spatial_scale * src_w * exp(dsx)
    //                                                                 = (pw + 0.5) / pooled_width * spatial_scale * src_w * exp(dsx)
    // dwdiff / dsx = dwdiff / dwe * dwe / dsx + dwdiff / dws * dws / dsx = 1 * ((pw+1)/pooled_width + 0.5) * spatial_scale * src_w * exp(dsx) + (-1) * (pw/pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)
    //                                                                    = (wend-wstart) >= 1 ? (1 / pooled_width + 1) * spatial_scale * src_w * exp(dsx) : 0
    // dhctr / dsy  = (ph + 0.5) / pooled_height * spatial_scale * src_h * exp(dsy)
    // dhdiff / dsy = (hend-hstart) >= 1 ? (1 / pooled_height + 1) * spatial_scale * src_h * exp(dsy) : 0
    
    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);         //  
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    //bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    //Dtype maxval = is_empty ? 0 : -FLT_MAX;
    Dtype val = 0; Dtype gain = 0, gain_x = 0, gain_y = 0;   
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    //int maxidx = -1;
    bottom_data += (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        Dtype w_ = w, h_ = h;  
        //if (bottom_data[bottom_index] > maxval) {
        //  maxval = bottom_data[bottom_index];
        //  maxidx = bottom_index;
        //}
        //gain_x = max(0., 1 - abs( dst_ctr_x + static_cast<Dtype>(pw) / static_cast<Dtype>(pooled_width) * dst_scl_x - w ));
        //gain_y = max(0., 1 - abs( dst_ctr_y + static_cast<Dtype>(ph) / static_cast<Dtype>(pooled_height) * dst_scl_y - h));
        gain_x = (wdiff - abs((w_ - wctr))) / wdiff;
        gain_y = (hdiff - abs((h_ - hctr))) / hdiff; 
        gain = gain_x * gain_y;
        //val = val + gain * bottom_data[bottom_index];
        //val = val + bottom_data[bottom_index];
        val = val + gain;
      }
    }
    //top_data[index] = maxval;
    //argmax_data[index] = maxidx;
    //top_data[index] = val;
    top_data[index] = ph; //static_cast<int>(floor(static_cast<Dtype>(ph) * bin_size_h)) + roi_start_h; 
  }
}

extern "C"
void inn_ROIWarping_updateOutput(THCState *state,
    THCudaTensor *output, /*THCudaTensor *indices,*/
    THCudaTensor *data, THCudaTensor* rois, THCudaTensor* delta_rois, int W, int H, double spatial_scale)
{
  THAssert(THCudaTensor_nDimension(state, data) == 4);
  THAssert(THCudaTensor_nDimension(state, rois) == 2 && rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, delta_rois) == 2 && delta_rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, rois) == THCudaTensor_nDimension(state, delta_rois) &&
           rois->size[0] == delta_rois->size[0] &&
           rois->size[1] == delta_rois->size[1]);
  THAssert(THCudaTensor_isContiguous(state, data));
  THAssert(THCudaTensor_isContiguous(state, rois));
  THAssert(THCudaTensor_isContiguous(state, delta_rois));
  long num_rois = rois->size[0];
  long nInputPlane = data->size[1];
  THCudaTensor_resize4d(state, output, num_rois, nInputPlane, H, W);
  //THCudaTensor_resize4d(state, indices, num_rois, nInputPlane, H, W);

  long count = THCudaTensor_nElement(state, output);

  ROIWarpForward<float><<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count,
      THCudaTensor_data(state, data),
      spatial_scale, nInputPlane, data->size[2], data->size[3], H, W,
      THCudaTensor_data(state, rois),
      THCudaTensor_data(state, delta_rois),
      THCudaTensor_data(state, output) /*,
      (int*)THCudaTensor_data(state, indices)*/
      );

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in inn_ROIWarping_updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}

//template <typename Dtype>
//__global__ void ROIWarpBackwardAtomic(const int nthreads, const Dtype* top_diff,
//    /*const int* argmax_data,*/ const int num_rois, const Dtype spatial_scale,
//    const int channels, const int height, const int width,
//    const int pooled_height, const int pooled_width, Dtype* bottom_diff,
//    const Dtype* bottom_rois) {
//  CUDA_KERNEL_LOOP(index, nthreads) {
//    // (n, c, ph, pw) is an element in the pooled output
//    int pw = index % pooled_width;
//    int ph = (index / pooled_width) % pooled_height;
//    int c = (index / pooled_width / pooled_height) % channels;
//    int n = index / pooled_width / pooled_height / channels;
//
//    bottom_rois += n * 5;
//    int roi_batch_ind = (bottom_rois[0] - 1);
//    int bottom_offset = (roi_batch_ind * channels + c) * height * width;
//    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
//    const Dtype* offset_top_diff = top_diff + top_offset;
//    Dtype* offset_bottom_diff = bottom_diff + bottom_offset;
//    //const int* offset_argmax_data = argmax_data + top_offset;
//
//    //int argmax = offset_argmax_data[ph*pooled_width + pw];
//    //if(argmax != -1) {
//    //  atomicAdd(offset_bottom_diff + argmax, offset_top_diff[ph * pooled_width + pw]);
//    //}
//  }
//}

template <typename Dtype>
__global__ void ROIWarpBackward(const int nthreads, /*const Dtype* bottom_data,*/
    const Dtype spatial_scale, const int channels, const int height, const int width, 
    const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois, const Dtype* bottom_delta_rois, 
    const Dtype* top_diff,
    Dtype* bottom_diff_data, 
    Dtype* bottom_diff_delta_rois_buffer/*,
    Dtype* bottom_diff_delta_rois_cx,
    Dtype* bottom_diff_delta_rois_cy,
    Dtype* bottom_diff_delta_rois_sx,
    Dtype* bottom_diff_delta_rois_sy*/) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = (bottom_rois[0] - 1);

    Dtype src_w = bottom_rois[3] - bottom_rois[1] + 1; 
    Dtype src_h = bottom_rois[4] - bottom_rois[2] + 1;
    Dtype src_ctr_x = bottom_rois[1] + 0.5*(src_w-1.0); 
    Dtype src_ctr_y = bottom_rois[2] + 0.5*(src_h-1.0); 

    Dtype dst_ctr_x = bottom_delta_rois[1]; // dx (in fast-rcnn notation) = cx (in here)
    Dtype dst_ctr_y = bottom_delta_rois[2]; // dy (in fast-rcnn notation) = cy (in here) 
    Dtype dst_scl_x = bottom_delta_rois[3]; // dw (in fast-rcnn notation) = sx (in here)
    Dtype dst_scl_y = bottom_delta_rois[4]; // dh (in fast-rcnn notation) = sy (in here) 

    Dtype pred_ctr_x = dst_ctr_x * src_w + src_ctr_x; // dpcx / dcx = src_w
    Dtype pred_ctr_y = dst_ctr_y * src_h + src_ctr_y; // dpcy / dcy = src_h
    Dtype pred_w = exp(dst_scl_x) * src_w;            // dpw  / dsx = src_w * exp(dsx)  
    Dtype pred_h = exp(dst_scl_y) * src_h;            // dph  / dsy = src_h * exp(dsy)  
    
    int roi_start_w = ( round(pred_ctr_x - 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drsw / dpcx = spatial_scale; drsw / dpw = -0.5 * spatial_scale
    int roi_start_h = ( round(pred_ctr_y - 0.5*(pred_h-1)) - 1 ) * spatial_scale; // drsh / dpcy = spatial_scale; drsh / dph = -0.5 * spatial_scale
    int roi_end_w =   ( round(pred_ctr_x + 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drew / dpcx = spatial_scale; drew / dpw =  0.5 * spatial_scale
    int roi_end_h =   ( round(pred_ctr_y + 0.5*(pred_h-1)) - 1 ) * spatial_scale; // dreh / dpcy = spatial_scale; dreh / dph =  0.5 * spatial_scale
    assert(roi_end_w - roi_start_w >= 0); 
    assert(roi_end_h - roi_start_h >= 0); 
    
    // drsw / dcx = drsw / dpcx * dpcx / dcx = spatial_scale * src_w
    // drew / dcx = drew / dpcx * dpcx / dcx = spatial_scale * src_w

    // drsh / dcy = drsh / dpcy * dpcy / dcy = spatial_scale * src_h
    // dreh / dcy = dreh / dpcy * dpcy / dcy = spatial_scale * src_h

    // drsw / dsx = drsw / dpw * dpw / dsx = -0.5 * spatial_scale * src_w * exp(dsx) 
    // drew / dsx = drew / dpw * dpw / dsx =  0.5 * spatial_scale * src_w * exp(dsx)
 
    // drsh / dsy = drsh / dph * dph / dsy = -0.5 * spatial_scale * src_h * exp(dsy)
    // dreh / dsy = dreh / dph * dph / dsy =  0.5 * spatial_scale * src_h * exp(dsy) 
 
    // Force malformed ROIs to be 1x1
    int roi_width  = roi_end_w - roi_start_w + 1;   // drw / drew = (rew - rsw) > 0 ?  1 : 0 
                                                    // drw / drsw = (rew - rsw) > 0 ? -1 : 0
    int roi_height = roi_end_h - roi_start_h + 1;   // drh / dreh = (reh - rsh) > 0 ?  1 : 0
                                                    // drh / drsh = (reh - rsh) > 0 ? -1 : 0
    // drw / dcx = drw / drew * drew / dcx + drw / drsw * drsw / dcx = drew / dcx - drsw / dcx = spatial_scale * src_w - spatial_scale * src_w = 0 
    // drh / dcy = drh / dreh * dreh / dcy + drh / drsh * drsh / dcy = dreh / dcy - drsh / dcy = spatial_scale * src_h - spatial_scale * src_h = 0 
    // drw / dsx = drw / drew * drew / dsx + drw / drsw * drsw / dsx = drew / dsx - drsw / dsx = 0.5 * spatial_scale * src_w * exp(dsx) - (-0.5 * spatial_scale * src_w * exp(dsx)) = spatial_scale * src_w * exp(dsx) 
    // drh / dsy = drh / dreh * dreh / dsy + drh / drsh * drsh / dsy = dreh / dsy - drsh / dsy = 0.5 * spatial_scale * src_h * exp(dsy) - (-0.5 * spatial_scale * src_h * exp(dsy)) = spatial_scale * src_h * exp(dsy) 

    Dtype bin_size_w = static_cast<Dtype>(roi_width)
                       / static_cast<Dtype>(pooled_width);  // dbw / drw  =  1 / pooled_width
    Dtype bin_size_h = static_cast<Dtype>(roi_height)         
                       / static_cast<Dtype>(pooled_height); // dbh / drh  =  1 / pooled_height
    // dbw / dcx = dbw / drw * drw / dcx = 0 
    // dbh / dcy = dbh / drh * drh / dcy = 0
    // dbw / dsx = dbw / drw * drw / dsx = 1 / pooled_width * spatial_scale * src_w * exp(dsx) 
    // dbh / dsy = dbh / drh * drh / dsy = 1 / pooled_height * spatial_scale * src_h * exp(dsy) 

    int wstart = static_cast<int>(floor(static_cast<Dtype>(pw)        // dws / dbw = pw 
                                        * bin_size_w)) + roi_start_w; // dws / drsw = 1
    int hstart = static_cast<int>(floor(static_cast<Dtype>(ph)        // dhs / dbh = ph 
                                        * bin_size_h)) + roi_start_h; // dhs / drsh = 1 
    int wend = static_cast<int>(ceil(static_cast<Dtype>(pw + 1)       // dwe / dbw = (pw+1)
                                     * bin_size_w)) + roi_start_w;    // dwe / drsw = 1 
    int hend = static_cast<int>(ceil(static_cast<Dtype>(ph + 1)       // dhe / dbh = (ph+1)
                                     * bin_size_h)) + roi_start_h;    // dhe / drsh = 1 
    // dws / dcx = dws / dbw * dbw / dcx + dws / drsw * drsw / dcx = pw * 0 + 1 * spatial_scale * src_w     = spatial_scale * src_w
    // dwe / dcx = dwe / dbw * dbw / dcx + dwe / drsw * drsw / dcx = (pw+1) * 0 + 1 * spatial_scale * src_w = spatial_scale * src_w

    // dws / dsx = dws / dbw * dbw / dsx + dws / drsw * drsw / dsx = pw * 1 / pooled_width * spatial_scale * src_w * exp(dsx) + 1 * (-0.5) * spatial_scale * src_w * exp(dsx) = ( pw / pooled_width - 0.5 ) * spatial_scale * src_w * exp(dsx) 
    // dwe / dsx = dwe / dbw * dbw / dsx + dwe / drsw * drsw / dsx = (pw+1) * 1 / pooled_width * spatial_scale * src_w * exp(dsx) + 1 * 0.5 * spatial_scale * src_w * exp(dsx) = ( (pw+1)/pooled_width + 0.5 ) * spatial_scale * src_w * exp(dsx)

    // dhs / dcy = dhs / dbh * dbh / dcy + dhs / drsh * drsh / dcy = ph * 0 + 1 * spatial_scale * src_h     = spatial_scale * src_w
    // dhe / dcy = dhe / dbh * dbh / dcy + dhe / drsh * drsh / dcy = (ph+1) * 0 + 1 * spatial_scale * src_h = spatial_scale * src_h

    // dhs / dsy = dhs / dbh * dbh / dsy + dhs / drsh * drsh / dsy = ph * 1 / pooled_height * spatial_scale * src_h * exp(dsy) + 1 * (-0.5) * spatial_scale * src_h * exp(dsy) = (ph / pooled_height - 0.5) * spatial_scale * src_h * exp(dsy) 
    // dhe / dsy = dhe / dbh * dbh / dsy + dhe / drsh * drsh / dsy = (ph+1) * 1 / pooled_height * spatial_scale * src_h * exp(dsy) + 1 * 0.5 * spatial_scale * src_h * exp(dsy) = ((ph+1)/pooled_height + 0.5) * spatial_scale * src_h * exp(dsy)  

    Dtype wctr = static_cast<Dtype>(wend-1+wstart) * 0.5;      // dwctr / dwe = 0.5; dwctr / dws = 0.5 
    Dtype hctr = static_cast<Dtype>(hend-1+hstart) * 0.5;      // dhctr / dhe = 0.5; dhctr / dhs = 0.5 
    Dtype wdiff = max(static_cast<Dtype>(wend-1-wstart), 1.);  // dwdiff / dwe = (wend-wstart) >= 1 ? 1 : 0; dwdiff / dws = (wend-wstart) >= 1 ? -1 : 0; 
    Dtype hdiff = max(static_cast<Dtype>(hend-1-hstart), 1.);  // dhdiff / dhe = (hend-hstart) >= 1 ? 1 : 0; dhdiff / dhs = (hend-hstart) >= 1 ? -1 : 0;
    Dtype wdiff_mask = (wend-wstart) >= 1 ? 1 : 0;
    Dtype hdiff_mask = (wend-wstart) >= 1 ? 1 : 0;
    // dwctr / dcx = dwctr / dwe * dwe / dcx + dwctr / dws * dws / dcx = 0.5 * spatial_scale * src_w + 0.5 * spatial_scale * src_w = spatial_scale * src_w 
    // dwdiff / dcx = dwdiff / dwe * dwe / dcx + dwdiff / dws * dws / dcx = 1 * spatial_scale * src_w -  1  * spatial_scale * src_w = 0 

    // dhctr / dcy = spatial_scale * src_h
    // dhdiff / dcy = 0
  
    // dwctr / dsx = dwctr / dwe * dwe / dsx + dwctr / dws * dws / dsx = 0.5 * ((pw+1)/pooled_width + 0.5) * spatial_scale * src_w * exp(dsx) + 0.5 * (pw/pooled_width - 0.5) * spatial_scale * src_w * exp(dsx) 
    //                                                                 = 0.5 * (2*pw+1)/pooled_width * spatial_scale * src_w * exp(dsx)
    //                                                                 = (pw + 0.5) / pooled_width * spatial_scale * src_w * exp(dsx) 
    // dwdiff / dsx = dwdiff / dwe * dwe / dsx + dwdiff / dws * dws / dsx = 1 * ((pw+1)/pooled_width + 0.5) * spatial_scale * src_w * exp(dsx) + (-1) * (pw/pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)
    //                                                                    = (wend-wstart) >= 1 ? (1 / pooled_width + 1) * spatial_scale * src_w * exp(dsx) : 0 
    // dhctr / dsy  = (ph + 0.5) / pooled_height * spatial_scale * src_h * exp(dsy)
    // dhdiff / dsy = (hend-hstart) >= 1 ? (1 / pooled_height + 1) * spatial_scale * src_h * exp(dsy) : 0
  
    // if w >= wctr  
    // dgx / dcx = dgx / dwctr * dwctr / dcx + dgx / dwdiff * dwdiff / dcx = 1 / wdiff * spatial_scale * src_w + (( w - wctr ) / (wdiff)^2 ) * 0
    //                                                                     = 1 / wdiff * spatial_scale * src_w  
    // dgx / dsx = dgx / dwctr * dwctr / dsx + dgx / dwdiff * dwdiff / dsx = 1 / wdiff * (pw + 0.5) / pooled_width * spatial_scale * src_w * exp(dsx) + ((wend-wstart) >= 1 ? 1 : 0) * (( w - wctr ) / (wdiff)^2 ) * (1 / pooled_width + 1) * spatial_scale * src_w * exp(dsx)
    //                                                                     = ((pw * 0.5) / (pooled_width * wdiff) + ((wend-wstart) >= 1 ? 1 : 0) * (( w - wctr ) / (wdiff)^2 ) * (1 + pooled_width) / pooled_width ) * spatial_scale * src_w * exp(dsx) 
    // dgy / dcy = dgy / dhctr * dhctr / dcy + dgy / dhdiff * dhdiff / dcy = 1 / hdiff * spatial_scale * src_h
    // dgy / dsy = dgy / dhctr * dhctr / dsy + dgy / dhdiff * dhdiff / dsy = ((ph * 0.5) / (pooled_height * hdiff) + ((hend-hstart) >= 1 ? 1 : 0) * (( h - hctr ) / (hdiff)^2 ) * (1 + pooled_height) / pooled_height ) * spatial_scale * src_h * exp(dsy) 
  
  
    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);         //  
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    //bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    Dtype val_cx = 0, val_cy = 0, val_sx = 0, val_sy = 0; 
    Dtype gain = 0, gain_x = 0, gain_y = 0;  
    Dtype pw_ = static_cast<Dtype>(pw); 
    Dtype ph_ = static_cast<Dtype>(ph);
    Dtype pooled_width_  = static_cast<Dtype>(pooled_width); 
    Dtype pooled_height_ = static_cast<Dtype>(pooled_height);
    Dtype src_w_ = static_cast<Dtype>(src_w); 
    Dtype src_h_ = static_cast<Dtype>(src_h);  
    Dtype buffer_sx = 0, buffer_sy = 0;  
    //bottom_data += (roi_batch_ind * channels + c) * height * width;
    bottom_diff_data += (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        Dtype w_ = w, h_ = h;  
        gain_x = (wdiff - abs((w_ - wctr))) / wdiff;   // dgx / dwdiff =   (w-wctr) / (wdiff)^2 ( if w >= wctr ) 
                                                       // dgx / dwdiff = - (w-wctr) / (wdiff)^2 ( else )
                                                       // dgx / dwctr  =   1 / wdiff ( if w >= wctr )  
                                                       // dgx / dwctr  = - 1 / wdiff ( else )  
        gain_y = (hdiff - abs((h_ - hctr))) / hdiff;   // dgy / dhdiff =   (h-hctr) / (hdiff)^2 ( if h >= hctr ) 
                                                                                              // dgy / dhdiff = - (h-hctr) / (hdiff)^2 ( else )
                                                                                              // dgy / dhctr  =   1 / hdiff ( if h >= hctr )
                                                                                              // dgy / dhdiff = - 1 / hdiff ( else )
        gain = gain_x * gain_y;
        //bottom_diff_data[bottom_index] = bottom_diff_data[bottom_index] + gain * top_diff[index]; //val = val + gain * bottom_data[bottom_index];
        bottom_diff_data[bottom_index] = ph; //static_cast<int>(floor(static_cast<Dtype>(ph) * bin_size_h)) + roi_start_h;

        // buffer 
        Dtype coeff_x = w >= wctr ? 1 : -1; coeff_x = coeff_x * gain_y * spatial_scale * src_w_ * top_diff[index]; 
        Dtype coeff_y = h >= hctr ? 1 : -1; coeff_y = coeff_y * gain_x * spatial_scale * src_h_ * top_diff[index]; 
        val_cx = val_cx + coeff_x / wdiff;  
        val_cy = val_cy + coeff_y / hdiff;
        //val_sx = val_sx + coeff_x * (pw_ * 0.5 * wdiff + (w_ - wctr) * (1 + pooled_width_ )) / (wdiff*wdiff) / pooled_width_  * exp(dst_scl_x);
        //val_sy = val_sy + coeff_y * (ph_ * 0.5 * hdiff + (h_ - hctr) * (1 + pooled_height_)) / (hdiff*hdiff) / pooled_height_ * exp(dst_scl_y);
        buffer_sx = 0; buffer_sx = coeff_x * (pw_ * 0.5 * wdiff + wdiff_mask * (w_ - wctr) * (1 + pooled_width_ )); buffer_sx = buffer_sx / (wdiff*wdiff); buffer_sx = buffer_sx / pooled_width_  * exp(dst_scl_x);  
        val_sx = val_sx + buffer_sx; 
        buffer_sy = 0; buffer_sy = coeff_y * (ph_ * 0.5 * hdiff + hdiff_mask * (h_ - hctr) * (1 + pooled_height_)); buffer_sy = buffer_sy / (hdiff*hdiff); buffer_sy = buffer_sy / pooled_height_ * exp(dst_scl_y);
        val_sy = val_sy + buffer_sy; 
        //(dgain/ddelta_rois) * top_diff[index]; // dgain/ddeleta_rois = dgain/dgain_x * dgain_x/ddelta_rois + dgain/dgain_y * dgain_y/ddelta_rois
                                                 //                    =        gain_y * dgain_x/ddelta_rois +        gain_x * dgain_y/ddelta_rois
      }
    }
    int buffer_index = n * (channels * pooled_height * pooled_width * 4) + c * (pooled_height * pooled_width * 4) + ph * (pooled_width * 4) + pw * 4; 
    bottom_diff_delta_rois_buffer[buffer_index+0] = val_cx; 
    bottom_diff_delta_rois_buffer[buffer_index+1] = val_cy; 
    bottom_diff_delta_rois_buffer[buffer_index+2] = val_sx;
    bottom_diff_delta_rois_buffer[buffer_index+3] = val_sy;
    //bottom_diff_delta_rois_cx[index] = val_cx;
    //bottom_diff_delta_rois_cy[index] = val_cy;
    //bottom_diff_delta_rois_sx[index] = val_sx;
    //bottom_diff_delta_rois_sy[index] = val_sy;
  }
}


extern "C"
void inn_ROIWarping_updateGradInputAtomic(THCState *state,
    THCudaTensor *gradInput_data, THCudaTensor *data,
    THCudaTensor *gradInput_delta_rois, THCudaTensor *delta_rois,
    THCudaTensor *gradInput_delta_rois_buffer,
    THCudaTensor *gradOutput, THCudaTensor* rois, int W, int H, double spatial_scale)
{
  THAssert(THCudaTensor_nDimension(state, data) == 4);
  THAssert(THCudaTensor_nDimension(state, rois) == 2 && rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, delta_rois) == 2 && delta_rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, rois) == THCudaTensor_nDimension(state, delta_rois) &&
           rois->size[0] == delta_rois->size[0] &&
           rois->size[1] == delta_rois->size[1]);
  THAssert(THCudaTensor_isContiguous(state, data));
  THAssert(THCudaTensor_isContiguous(state, rois));
  THAssert(THCudaTensor_isContiguous(state, delta_rois));
  long num_rois = rois->size[0];
  long nInputPlane = data->size[1];
  THCudaTensor_resizeAs(state, gradInput_data, data);
  THCudaTensor_zero(state, gradInput_data);
  THCudaTensor_resizeAs(state, gradInput_delta_rois, delta_rois);
  THCudaTensor_zero(state, gradInput_delta_rois);

  THCudaTensor_resize5d(state, gradInput_delta_rois_buffer, num_rois, nInputPlane, H, W, 4);
  THCudaTensor_zero(state, gradInput_delta_rois_buffer);
  //THCudaTensor_resizeAs(state, gradInput_delta_rois_dx, gradOutput);
  //THCudaTensor_zero(state, gradInput_delta_rois_dx);
  //THCudaTensor_resizeAs(state, gradInput_delta_rois_dy, gradOutput);
  //THCudaTensor_zero(state, gradInput_delta_rois_dy);
  //THCudaTensor_resizeAs(state, gradInput_delta_rois_dw, gradOutput);
  //THCudaTensor_zero(state, gradInput_delta_rois_dw);
  //THCudaTensor_resizeAs(state, gradInput_delta_rois_dh, gradOutput);
  //THCudaTensor_zero(state, gradInput_delta_rois_dh);

  long count = THCudaTensor_nElement(state, gradOutput);

  //ROIWarpBackwardAtomic<float><<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
  //    count,
  //    THCudaTensor_data(state, gradOutput),
  //    //(int*)THCudaTensor_data(state, indices),
  //    num_rois, spatial_scale, nInputPlane, data->size[2], data->size[3], H, W,
  //    THCudaTensor_data(state, gradInput_data),
  //    THCudaTensor_data(state, rois)
  //    );
  ROIWarpBackward<float><<<GET_BLOCKS(count), CUDA_NUM_THREADS / 2, 0, THCState_getCurrentStream(state)>>>(
      count,
      //THCudaTensor_data(state, data),
      spatial_scale, nInputPlane, data->size[2], data->size[3], H, W,
      THCudaTensor_data(state, rois),
      THCudaTensor_data(state, delta_rois),
      THCudaTensor_data(state, gradOutput), 
      THCudaTensor_data(state, gradInput_data),
      THCudaTensor_data(state, gradInput_delta_rois_buffer)/*,
      THCudaTensor_data(state, gradInput_delta_rois_dx),
      THCudaTensor_data(state, gradInput_delta_rois_dy),
      THCudaTensor_data(state, gradInput_delta_rois_dw),
      THCudaTensor_data(state, gradInput_delta_rois_dh)*/
      );

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in inn_ROIWarping_updateGradInputAtomic: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}
